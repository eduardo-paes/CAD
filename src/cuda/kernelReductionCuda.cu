
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <stdlib.h>
#include <time.h>
#include <omp.h>
#include <sys/time.h>

double T()
{
   struct timeval tv;
   gettimeofday(&tv, NULL);

   return (tv.tv_sec * 1000 * 1000 + tv.tv_usec) / 1000000.;
}

#define GCE(ans) {gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true)
{
    if (code != hipSuccess)
    {
        fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort)
            exit(code);
    }
}

#define BLOCKSIZE 512
__global__ void vector_sum(double *x, double *output, int n)
{

   int gid = blockIdx.x * blockDim.x + threadIdx.x;
   int lid = threadIdx.x;
   int blockSize = blockDim.x;

   __shared__ double partial_sum[BLOCKSIZE];

   partial_sum[lid] = gid < n ? x[gid] : 0.0;

   __syncthreads(); // block barrier

   for (int i = blockSize / 2; i > 0; i /= 2)
   {
      if (lid < i)
      {
         partial_sum[lid] += partial_sum[lid + i];
      }
      __syncthreads();
   }

   if (lid == 0)
   {
      output[blockIdx.x] = partial_sum[0];
   }
}

__global__ void vector_sum_global(double *x, double *output, int n)
{

   int gid = blockIdx.x * blockDim.x + threadIdx.x;
   int lid = threadIdx.x;
   int block_size = blockDim.x;

   __syncthreads(); // block barrier

   for (int i = block_size / 2; i > 0; i /= 2)
   {
      if (lid < i)
      {
         x[gid] += x[gid + i];
      }
      __syncthreads();
   }

   if (lid == 0)
   {
      output[blockIdx.x] = x[blockIdx.x * blockDim.x];
   }
}

int main(int argc, char *argv[])
{

   int n = 102400000;
   int threadsPerBlock = 512;
   int numberOfBlocks = (n + threadsPerBlock - 1) / threadsPerBlock;

   double *h_x;
   double *h_out;

   double *d_x;
   double *d_out;

   double T0, T1;
   size_t size_x = n * sizeof(double);
   size_t size_out = numberOfBlocks * sizeof(double);

   h_x = (double *)malloc(size_x);
   h_out = (double *)malloc(size_out);

   // Allocate memory for each vector on GPU
   GCE(hipMalloc(&d_x, size_x));
   GCE(hipMalloc(&d_out, size_out));

   int i;
   for (i = 0; i < n; i++)
   {
      h_x[i] = 1. / n; //
   }

   T0 = T();
   GCE(hipMemcpy(d_x, h_x, size_x, hipMemcpyHostToDevice));
   T1 = T();
   printf("SEND  %.6f\n", T1 - T0);

   vector_sum<<<numberOfBlocks, threadsPerBlock>>>(d_x, d_out, n);
   // vector_sum_global<<<numberOfBlocks, threadsPerBlock>>>(d_x, d_out,n );

   T0 = T();
   GCE(hipMemcpy(h_out, d_out, size_out, hipMemcpyDeviceToHost));
   T1 = T();
   printf("RETURN %.6f\n", T1 - T0);

   double sum = 0;

   for (i = 0; i < numberOfBlocks; i++)
   {
      sum += h_out[i];
      // printf("partial: %lf\n", h_out[i] );
   }

   printf("final result: %f\n", sum);
   hipFree(d_x);
   hipFree(d_out);

   free(h_x);
   free(h_out);

   return 0;
}
